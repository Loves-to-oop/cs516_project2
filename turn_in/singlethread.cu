#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

/*
#include "cuPrintf.cu"`
 */


using namespace std;

inline void __cudaSafeCall( hipError_t err,
		const char *file, const int line ) 

{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{

			fprintf( stderr,
					"cudaSafeCall() failed at %s:%i : %s\n",
					file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );



#pragma warning( pop ) 
#endif
	// CUDA_CHECK_ERROR

	return;
}//end function

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{

		hipError_t err = hipGetLastError();	
		if( hipSuccess != err )
		{
			fprintf( stderr,
					"cudaCheckError() with sync failed at %s:%i : %s.\n", 
					file, line, hipGetErrorString( err ) );
			exit( -1 );


		}

		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{

			if ( hipSuccess != err )
				fprintf( stderr,
						"cudaCheckError() failed at %s:%i : %s.\n",
						file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );


	// More careful checking. However, this will affect performance. // Comment if not needed

#pragma warning( pop )
#endif // CUDA_CHECK_ERROR

	return;

}

void bubble_sort(int * array, int size)
{


	for(int i = 0; i <= size - 1; i ++)
	{

		for(int j = 1; j <= size - 1; j ++)
		{


			if(array[j] <  array[j - 1])
			{

				//printf("%d %d\n", array[j - 1], array[j]);

				int c = array[j - 1];

				array[j - 1] = array[j];

				array[j] = c;

				//printf("%d %d\n\n", array[j - 1], array[j]);

			}//end if




		}//end for j

	}//end for i


}//end function

void print_array(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function


int * makeRandArray( const int size, const int seed ) {
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array; }


	/*

	   Kernel is fuction to run on GPU.

	 */

	__global__ void matavgKernel(int * array, int size ) {

		//array[0] = 5;
		for(int i = 0; i <= size - 1; i ++)
		{


			//cuPrintf(“Value is: %d\n”, i);

			for(int j = 1; j <= size - 1; j ++)
			{


				if(array[j] <  array[j - 1])
				{

					//printf("%d %d\n", array[j - 1], array[j]);

					int c = array[j - 1];

					array[j - 1] = array[j];

					array[j] = c;

					//printf("%d %d\n\n", array[j - 1], array[j]);

				}//end if




			}//end for j

		}//end for i


		//return array;

	}//end function


void check_sorted(int * host_array, int * array, int size)
{

	for(int i = 1; i <= size - 1; i ++)
	{

		//printf("%d >= %d\n", host_array[i], host_array[i - 1]);

		assert(host_array[i] >= host_array[i - 1]); 

	}//end for i

	printf("sorted order\n");

	for(int i = 0; i <= size - 1; i ++)
	{
		int missing_number = 1;

		//		printf("checking: %d, ", array[i]);

		for(int j = 0; j <= size - 1; j ++)
		{

			if(array[i] == host_array[j])
			{

				//				printf("FOUND\n");

				missing_number = 0;

			}//end if

		}//end for j

		assert(missing_number == 0);

	}//end for i

	printf("none missing\n");

}//end function


int main( int argc, char* argv[] ) {
	int * array; // the poitner to the array of rands 
	int size, seed; // values for the size of the array 
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 3 ){
		std::cerr << "usage: "
			<< argv[0]
			<< " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]"
			<< std::endl;
		exit( -1 ); }
	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	} {
		std::stringstream ss1( argv[2] ); 
		ss1 >> seed; }
	
	// get the random numbers
	array = makeRandArray( size, seed );

	int * host_array = (int*)malloc(size * 4);

	for(int i =0; i <= size - 1; i ++)
	{
	
		host_array[i] = array[i];
	
	}//end for i

	//print_array(array, size);

//printf("host_array\n");

//print_array(host_array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////


	int * cuda_array;

	hipMalloc(&cuda_array, size * 4);

	hipMemcpy(cuda_array, host_array, size * 4, hipMemcpyHostToDevice);


	matavgKernel <<< 1, 1 >>> (cuda_array, size); 

	hipMemcpy(host_array, cuda_array, size * 4, hipMemcpyDeviceToHost);

	hipFree(cuda_array);

	//https://stackoverflow.com/questions/6419700/way-to-verify-kernel-was-executed-in-cuda

	/***********************************
	 *
	 Stop and destroy the cuda timer
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	  end of cuda timer destruction
	 **********************************/
	

if(argc == 5)
{

	check_sorted(host_array, array, size);

}//end if

if(argc == 6)
{

	print_array(host_array, size);

}//end if

if(argc != 4)
{
	
	std::cerr << "Total time in seconds: "
		<< timeTotal / 1000.0 << std::endl;
	printSorted = true;

	if( printSorted ){


		///////////////////////////////////////////////
		/// Your code to print the sorted array here //
		///////////////////////////////////////////////
	}//end if 
}//end if

}//end main
