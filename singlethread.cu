#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>


using namespace std;

inline void __cudaSafeCall( hipError_t err,
		const char *file, const int line ) 

{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{

			fprintf( stderr,
					"cudaSafeCall() failed at %s:%i : %s\n",
					file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );



#pragma warning( pop ) 
#endif
	// CUDA_CHECK_ERROR

	return;
}//end function

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{

		hipError_t err = hipGetLastError();	
		if( hipSuccess != err )
		{
			fprintf( stderr,
					"cudaCheckError() with sync failed at %s:%i : %s.\n", 
					file, line, hipGetErrorString( err ) );
			exit( -1 );


		}

		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{

			if ( hipSuccess != err )
				fprintf( stderr,
						"cudaCheckError() failed at %s:%i : %s.\n",
						file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );


	// More careful checking. However, this will affect performance. // Comment if not needed

#pragma warning( pop )
#endif // CUDA_CHECK_ERROR

	return;

}

void bubble_sort(int * array, int size)
{


	for(int i = 0; i <= size - 1; i ++)
	{

		for(int j = 1; j <= size - 1; j ++)
		{


			if(array[j] <  array[j - 1])
			{

//printf("%d %d\n", array[j - 1], array[j]);

				int c = array[j - 1];

				array[j - 1] = array[j];

				array[j] = c;

//printf("%d %d\n\n", array[j - 1], array[j]);

			}//end if




		}//end for j

	}//end for i


}//end function

void print_array(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function


int * makeRandArray( const int size, const int seed ) {
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array; }


/*

   Kernel is fuction to run on GPU.

   */

	__global__ void matavgKernel(int * array, int size ) {


	for(int i = 0; i <= size - 1; i ++)
	{

		for(int j = 1; j <= size - 1; j ++)
		{


			if(array[j] <  array[j - 1])
			{

//printf("%d %d\n", array[j - 1], array[j]);

				int c = array[j - 1];

				array[j - 1] = array[j];

				array[j] = c;

//printf("%d %d\n\n", array[j - 1], array[j]);

			}//end if




		}//end for j

	}//end for i




	}

int main( int argc, char* argv[] ) {
	int * array; // the poitner to the array of rands 
	int size, seed; // values for the size of the array 
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 3 ){
		std::cerr << "usage: "
			<< argv[0]
			<< " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]"
			<< std::endl;
		exit( -1 ); }
	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	} {
		std::stringstream ss1( argv[2] ); 
		ss1 >> seed; }
	/*
	   {
	   int sortPrint;
	   std::stringstream ss1( argv[2] ); 
	   ss1 >> sortPrint;
	   if( sortPrint == 1 )
	   printSorted = true;
	   }
	 */
	// get the random numbers
	array = makeRandArray( size, seed );

	//print_array(array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////

hiprandState* devRandomGeneratorStateArray;
    hipMalloc ( &devRandomGeneratorStateArray, 1*sizeof( hiprandState ) );

	//bubble_sort(array, size);

	matavgKernel <<< 1, 1 >>> (array, size); 

	//matavgKerenel(array, size);

	/***********************************
	 *
	 Stop and destroy the cuda timer
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	  end of cuda timer destruction
	 **********************************/
	std::cerr << "Total time in seconds: "
		<< timeTotal / 1000.0 << std::endl;
	printSorted = true;

	if( printSorted ){


		for(int i = 0; i <= size - 1; i ++)
		{

			printf("%d, ", array[i]);

		}//end for i

		printf("\n");

		///////////////////////////////////////////////
		/// Your code to print the sorted array here //
		///////////////////////////////////////////////
	} }
