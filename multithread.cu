#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

/*
#include "cuPrintf.cu"`
 */

/*

   Working on using a 1D array to store the 2D buckets.
   Working in the kernel on calculating the start and finish
   in the 1D array to pass into the bubble sort function
   for each bucket.

   Figure out why bubble sort is not working on subarrays.

 */


using namespace std;

inline void __cudaSafeCall( hipError_t err,
		const char *file, const int line ) 

{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{

			fprintf( stderr,
					"cudaSafeCall() failed at %s:%i : %s\n",
					file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );



#pragma warning( pop ) 
#endif
	// CUDA_CHECK_ERROR

	return;
}//end function

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{

		hipError_t err = hipGetLastError();	
		if( hipSuccess != err )
		{
			fprintf( stderr,
					"cudaCheckError() with sync failed at %s:%i : %s.\n", 
					file, line, hipGetErrorString( err ) );
			exit( -1 );


		}

		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{

			if ( hipSuccess != err )
				fprintf( stderr,
						"cudaCheckError() failed at %s:%i : %s.\n",
						file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );


	// More careful checking. However, this will affect performance. // Comment if not needed

#pragma warning( pop )
#endif // CUDA_CHECK_ERROR

	return;

}

__device__ void bubble_sort(int * array, int size, int start, int finish)
{

	if((finish - start) == 1 && array[finish] < array[start])
	{

		printf("swap: %d, %d\n", array[start], array[finish]);

		int d = array[start];

		array[start] = array[finish];

		array[finish] = d;

	}//end if

	if((finish - start) > 1)
	{

		printf("%d - %d > 1 \n", finish, start);

		for(int i = start; i <= finish; i ++)
		{

			printf("i: %d\n", i);

			for(int j = start + 1; j <= finish; j ++)
			{


				if(array[j] <  array[j - 1])
				{

					printf("swap bubble: %d %d\n", array[j - 1], array[j]);

					int c = array[j - 1];

					array[j - 1] = array[j];

					array[j] = c;

					//printf("%d %d\n\n", array[j - 1], array[j]);

				}//end if




			}//end for j

		}//end for i

	}//end if

}//end function

void print_array(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function


__device__ void print_array_device(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function

int * makeRandArray( const int size, const int seed ) {
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array; }


	/*

	   Kernel is fuction to run on GPU.

	 */



	__global__ void matavgKernel(int * array, int size, int blocks_on_a_side, 
			int number_of_threads, int *array_of_buckets, int array_size, int * bucket_counts,
			int * bucket_starts, int * bucket_finishes) {

		//printf("blockdim.x: %d\n", blockDim.x);


		//i is what number, j is what digit to sort, then sort based on digit..

		int i = threadIdx.x + blockDim.x * blockIdx.x;
		int j = threadIdx.y + blockDim.y * blockIdx.y;

		int threads_on_a_side = (blockDim.x * blocks_on_a_side);

		int current = i + (j * threads_on_a_side);

		printf("%d = %d + (%d * %d)\n", current, i, j, threads_on_a_side);

		printf("current bucket size: %d\n", bucket_counts[current]);

		int bucket = 0;

		int start = 0;

		int finish = 0;

		bool start_set = false;

		bool finish_set = false;

		printf("i: %d, j: %d, current: %d, start: %d, finish: %d, bucket_start: %d, bucket_finish: %d\n", i, j, current, start, finish, bucket_starts[current],
				bucket_finishes[current]);

		if(bucket_starts[current] != -1)
		{
		
		bubble_sort(array_of_buckets, size, bucket_starts[current], bucket_finishes[current]);

		}//end if


	}//end function


int find_max_significant_digit(int * array, int size)
{






	return 0;

}//end function


void print_array_(int * host_array, int size)
{

	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", host_array[i]);

	}//end for i

	printf("\n");

}//end function

void unit_test()
{
}//end function


int main( int argc, char* argv[] ) {
	int * array; // the poitner to the array of rands 
	int size, seed; // values for the size of the array 
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 3 ){
		std::cerr << "usage: "
			<< argv[0]
			<< " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]"
			<< std::endl;
		exit( -1 ); }
	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	} {
		std::stringstream ss1( argv[2] ); 
		ss1 >> seed; }

	//unit_test();

	array = makeRandArray( size, seed );

	int * host_array = (int*)malloc(size * 10);

	for(int i =0; i <= size - 1; i ++)
	{

		host_array[i] = array[i];

	}//end for i

	print_array(array, size);

	printf("host_array\n");

	print_array(host_array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////

	int * cuda_array;

	hipMalloc(&cuda_array, size * 4);

	hipMemcpy(cuda_array, host_array, size * 4, hipMemcpyHostToDevice);

	int total_threads = (size / 10);

	if(total_threads > 48)
	{
	//	total_threads = 48;
	}//end if

	int diameter = sqrt(total_threads) + 1;

	printf("total threads: %d, diameter: %d\n", total_threads, diameter);

	int number_of_digits = 32;

	int threads_on_a_side = diameter / 5;

	printf("threads_on_a_side: %d\n", threads_on_a_side);

printf("threads per block: %f\n", pow(threads_on_a_side, 2));

	int blocks_on_a_side = (diameter / threads_on_a_side) + 1;

	printf("blocks_on_a_side: %d\n", blocks_on_a_side);


printf("blocks_per_grid: %f\n", pow(blocks_on_a_side, 2));

	int number_of_threads = pow(blocks_on_a_side * threads_on_a_side, 2);
	int number_of_buckets = number_of_threads;

	printf("number of threads: %d, buckets: %d\n", number_of_threads, number_of_buckets);

	dim3 threadsPerBlock(threads_on_a_side, threads_on_a_side);

	dim3 numBlocks(blocks_on_a_side, blocks_on_a_side);

	int ** array_of_buckets = new int*[number_of_buckets];

	int *bucket_counts = new int[number_of_buckets];


	int bucket_memory = 10000;

	for(int i = 0; i <= number_of_buckets - 1; i ++)
	{

		array_of_buckets[i] = new int[bucket_memory];


	}//end for i

	int max_value = 0;


	for(int i = 0; i <= size - 1; i ++)
	{


		if(array[i] > max_value)
			max_value = array[i];


	}//end for i

	printf("max: %d\n", max_value);

	for(int i = 0; i <= size - 1; i ++)
	{

		int bucket = ((double)array[i] / (double)(max_value + 1)) * number_of_buckets;

		printf("array[i]: %d, bucket: %d, ", array[i], bucket);

		printf("array[i] / max_value: %f, ", (double)array[i] / (double)(max_value + 1)); 

		array_of_buckets[bucket][bucket_counts[bucket]] = array[i]; 

		printf("value_in_array: %d, ", array_of_buckets[bucket][bucket_counts[bucket]]);

		bucket_counts[bucket] ++;

		printf("bucket count: %d, %d\n", 
				bucket_counts[bucket], 
				array_of_buckets[bucket][bucket_counts[bucket] - 1]);


	}//end for i


	int * cuda_bucket_counts;

	hipMalloc(&cuda_bucket_counts, number_of_buckets * 4);

	hipMemcpy(cuda_bucket_counts, bucket_counts, number_of_buckets * 4, hipMemcpyHostToDevice);

	size_t array_of_buckets_1D_size = size * 10;

	int * array_of_buckets_1D = new int[array_of_buckets_1D_size];

	int iter = 0;

	int *bucket_starts = new int[number_of_threads * 2];

	int *bucket_finishes = new int[number_of_threads * 2];

	int curr_bucket = 0;

	int curr_bucket2 = 0;

	for(int i = 0; i <= number_of_buckets - 1; i++)
	{

bucket_starts[curr_bucket] = -1;

bucket_finishes[curr_bucket2] = -1;

		for(int j = 0; j <= bucket_counts[i] - 1; j++)
		{

			if(j == 0)
			{

				bucket_starts[curr_bucket] = iter;

				printf("bucket_starts[%d] = %d\n", curr_bucket, iter);

				//	curr_bucket ++;

			}//end if

			if(j == bucket_counts[i] - 1)
			{

				bucket_finishes[curr_bucket2] = iter;

				printf("bucket_finishes[%d] = %d\n", curr_bucket2, iter);

				//curr_bucket2 ++;


			}//end if


			array_of_buckets_1D[iter] = array_of_buckets[i][j];

			iter ++;

		}//end for j

		curr_bucket ++;

		curr_bucket2 ++;

		array_of_buckets_1D[iter] = -1;

		iter ++;


	}//end for i

	for(int i = 0; i <= iter - 1; i ++)
	{

		printf("%d, ", array_of_buckets_1D[i]);


	}//end for i


	int * cuda_array_of_buckets;

int * cuda_bucket_starts;

int * cuda_bucket_finishes;

hipMalloc(&cuda_bucket_starts, number_of_threads * 10);

hipMemcpy(cuda_bucket_starts, bucket_starts, number_of_threads * 10,
		hipMemcpyHostToDevice);


hipMalloc(&cuda_bucket_finishes, number_of_threads * 10);

hipMemcpy(cuda_bucket_finishes, bucket_finishes, number_of_threads * 10,
		hipMemcpyHostToDevice);


	hipMalloc(&cuda_array_of_buckets, array_of_buckets_1D_size);

	hipMemcpy(cuda_array_of_buckets, array_of_buckets_1D, array_of_buckets_1D_size
			, hipMemcpyHostToDevice);

	matavgKernel <<< numBlocks, threadsPerBlock >>> 
		(cuda_array, size, blocks_on_a_side, 
		 number_of_threads, cuda_array_of_buckets, iter, cuda_bucket_counts, cuda_bucket_starts,
		 cuda_bucket_finishes); 

	hipMemcpy(array_of_buckets_1D, cuda_array_of_buckets, array_of_buckets_1D_size, hipMemcpyDeviceToHost);

	hipFree(cuda_array_of_buckets);

	printf("after sort:\n");

	print_array(array_of_buckets_1D, iter);

	hipMemcpy(host_array, cuda_array, size * 4, hipMemcpyDeviceToHost);

	hipFree(cuda_array);

	int j = 0;

	for(int i = 0; i <= iter - 1; i ++)
	{

		if(array_of_buckets_1D[i] != -1)
		{

			host_array[j] = array_of_buckets_1D[i];

			j++;

		}//end if

	}//end for i

	//https://stackoverflow.com/questions/6419700/way-to-verify-kernel-was-executed-in-cuda

	/***********************************
	 *
	 Stop and destroy the cuda timer
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	  end of cuda timer destruction
	 **********************************/
	std::cerr << "Total time in seconds: "
		<< timeTotal / 1000.0 << std::endl;
	printSorted = true;

	for(int i = 1; i <= size - 1; i ++)
	{

printf("%d >= %d\n", host_array[i], host_array[i - 1]);

		assert(host_array[i] >= host_array[i - 1]); 

	}//end for i

	for(int i = 0; i <= size - 1; i ++)
	{
		int missing_number = 1;

		printf("checking: %d, ", array[i]);

		for(int j = 0; j <= size - 1; j ++)
		{

			if(array[i] == host_array[j])
			{

				printf("FOUND\n");

				missing_number = 0;

			}//end if

		}//end for j

		assert(missing_number == 0);

	}//end for i


	if( printSorted ){

		print_array_(host_array, size);

		///////////////////////////////////////////////
		/// Your code to print the sorted array here //
		///////////////////////////////////////////////
	}//end if 


}//end main
