#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

/*
#include "cuPrintf.cu"`
 */


using namespace std;

inline void __cudaSafeCall( hipError_t err,
		const char *file, const int line ) 

{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{

			fprintf( stderr,
					"cudaSafeCall() failed at %s:%i : %s\n",
					file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );



#pragma warning( pop ) 
#endif
	// CUDA_CHECK_ERROR

	return;
}//end function

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{

		hipError_t err = hipGetLastError();	
		if( hipSuccess != err )
		{
			fprintf( stderr,
					"cudaCheckError() with sync failed at %s:%i : %s.\n", 
					file, line, hipGetErrorString( err ) );
			exit( -1 );


		}

		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{

			if ( hipSuccess != err )
				fprintf( stderr,
						"cudaCheckError() failed at %s:%i : %s.\n",
						file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );


	// More careful checking. However, this will affect performance. // Comment if not needed
#pragma warning( pop )
#endif // CUDA_CHECK_ERROR

	return;

}

void bubble_sort(int * array, int size)
{


	for(int i = 0; i <= size - 1; i ++)
	{

		for(int j = 1; j <= size - 1; j ++)
		{


			if(array[j] <  array[j - 1])
			{

				//printf("%d %d\n", array[j - 1], array[j]);

				int c = array[j - 1];

				array[j - 1] = array[j];

				array[j] = c;

				//printf("%d %d\n\n", array[j - 1], array[j]);

			}//end if




		}//end for j

	}//end for i


}//end function

void print_array(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function


int * makeRandArray( const int size, const int seed ) {
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array; }


	/*

	   Kernel is fuction to run on GPU.

	 */



	__device__ void bubble_sort_cuda(int * array, int size ) {

		//array[0] = 5;
		for(int i = 0; i <= size - 1; i ++)
		{


			//cuPrintf(“Value is: %d\n”, i);

			for(int j = 1; j <= size - 1; j ++)
			{


				if(array[j] <  array[j - 1])
				{

					//printf("%d %d\n", array[j - 1], array[j]);

					int c = array[j - 1];

					array[j - 1] = array[j];

					array[j] = c;

					//printf("%d %d\n\n", array[j - 1], array[j]);

				}//end if




			}//end for j

		}//end for i


		//return array;

	}//end function

__global__ void matavgKernel(int * array, int size ) {


	bubble_sort_cuda(array, size);


}//end function

int main( int argc, char* argv[] ) {
	int * array; // the poitner to the array of rands 
	int size, seed; // values for the size of the array 
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 3 ){
		std::cerr << "usage: "
			<< argv[0]
			<< " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]"
			<< std::endl;
		exit( -1 ); }
	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	} {
		std::stringstream ss1( argv[2] ); 
		ss1 >> seed; }
	/*
	   {
	   int sortPrint;
	   std::stringstream ss1( argv[2] ); 
	   ss1 >> sortPrint;
	   if( sortPrint == 1 )
	   printSorted = true;
	   }
	 */
	// get the random numbers

	array = makeRandArray( size, seed );

	int * host_array = (int*)malloc(size * 4);

	for(int i =0; i <= size - 1; i ++)
	{

		host_array[i] = array[i];

	}//end for i

	print_array(array, size);

	printf("host_array\n");

	print_array(host_array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////

	//hiprandState* devRandomGeneratorStateArray;
	//  hipMalloc ( &devRandomGeneratorStateArray, 1*sizeof( hiprandState ) );

	//bubble_sort(array, size);

	//    thrust::host_vector<int> hostCounts(1,  0);
	//  thrust::device_vector<int> deviceCounts(hostCounts);

	int * cuda_array;

	hipMalloc(&cuda_array, size * 4);

	hipMemcpy(cuda_array, host_array, size * 4, hipMemcpyHostToDevice);

	//matavgKernel <<< 1, 1 >>> (array, size); 

	matavgKernel <<< 1, 1 >>> (cuda_array, size); 

	hipMemcpy(host_array, cuda_array, size * 4, hipMemcpyDeviceToHost);

	hipFree(cuda_array);

	//https://stackoverflow.com/questions/6419700/way-to-verify-kernel-was-executed-in-cuda
	/*
	   hipError_t err = hipGetLastError();
	   if (err != hipSuccess) 
	   printf("Error: %s\n", hipGetErrorString(err));

	//thrust::reduce(deviceCounts.begin(), deviceCounts.end(), 0, thrust::plus<int>());;
	 */
	//matavgKerenel(array, size);

	/***********************************
	 *
	 Stop and destroy the cuda timer
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	  end of cuda timer destruction
	 **********************************/
	std::cerr << "Total time in seconds: "
		<< timeTotal / 1000.0 << std::endl;
	printSorted = true;

	if( printSorted ){


		for(int i = 0; i <= size - 1; i ++)
		{

			printf("%d, ", host_array[i]);

		}//end for i

		printf("\n");

		///////////////////////////////////////////////
		/// Your code to print the sorted array here //
		///////////////////////////////////////////////
	} }
