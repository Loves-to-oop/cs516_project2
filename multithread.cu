#include "hip/hip_runtime.h"

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

/*
#include "cuPrintf.cu"`
 */

/*

   Working on using a 1D array to store the 2D buckets.
   Working in the kernel on calculating the start and finish
   in the 1D array to pass into the bubble sort function
   for each bucket.

Figure out why bubble sort is not working on subarrays.

 */


using namespace std;

inline void __cudaSafeCall( hipError_t err,
		const char *file, const int line ) 

{
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{
		if ( hipSuccess != err )
		{

			fprintf( stderr,
					"cudaSafeCall() failed at %s:%i : %s\n",
					file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );



#pragma warning( pop ) 
#endif
	// CUDA_CHECK_ERROR

	return;
}//end function

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_CHECK_ERROR
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
	do
	{

		hipError_t err = hipGetLastError();	
		if( hipSuccess != err )
		{
			fprintf( stderr,
					"cudaCheckError() with sync failed at %s:%i : %s.\n", 
					file, line, hipGetErrorString( err ) );
			exit( -1 );


		}

		err = hipDeviceSynchronize();
		if( hipSuccess != err )
		{

			if ( hipSuccess != err )
				fprintf( stderr,
						"cudaCheckError() failed at %s:%i : %s.\n",
						file, line, hipGetErrorString( err ) );
			exit( -1 );

		}
	} while ( 0 );


	// More careful checking. However, this will affect performance. // Comment if not needed

#pragma warning( pop )
#endif // CUDA_CHECK_ERROR

	return;

}

__device__ void bubble_sort(int * array, int size, int start, int finish)
{

if((finish - start) == 1 && array[finish] < array[start])
{

printf("swap: %d, %d\n", array[start], array[finish]);

int d = array[start];

array[start] = array[finish];

array[finish] = d;

}//end if

	if((finish - start) > 1)
	{

printf("%d - %d > 1 \n", finish, start);

		for(int i = start; i <= finish; i ++)
		{

printf("i: %d\n", i);

			for(int j = start + 1; j <= finish; j ++)
			{


				if(array[j] <  array[j - 1])
				{

					printf("swap bubble: %d %d\n", array[j - 1], array[j]);

					int c = array[j - 1];

					array[j - 1] = array[j];

					array[j] = c;

					//printf("%d %d\n\n", array[j - 1], array[j]);

				}//end if




			}//end for j

		}//end for i

	}//end if

}//end function

void print_array(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function


__device__ void print_array_device(int * array, int size)
{



	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", array[i]);

	}//end for i

	printf("\n");



}//end function

int * makeRandArray( const int size, const int seed ) {
	srand( seed );
	int * array = new int[ size ];
	for( int i = 0; i < size; i ++ ) {
		array[i] = std::rand() % 1000000;
	}
	return array; }


	/*

	   Kernel is fuction to run on GPU.

	 */



	__global__ void matavgKernel(int * array, int size, int blocks_on_a_side, 
			int number_of_threads, int *array_of_buckets, int array_size, int * bucket_counts) {

		//printf("blockdim.x: %d\n", blockDim.x);


		//i is what number, j is what digit to sort, then sort based on digit..

		int i = threadIdx.x + blockDim.x * blockIdx.x;
		int j = threadIdx.y + blockDim.y * blockIdx.y;

		int threads_on_a_side = (blockDim.x * blocks_on_a_side);

		int current = i + (j * threads_on_a_side);

		printf("%d = %d + (%d * %d)\n", current, i, j, threads_on_a_side);

		printf("current bucket size: %d\n", bucket_counts[current]);

		int bucket = 0;

		int start = 0;

		int finish = 0;

		bool start_set = false;

		bool finish_set = false;

		for(int i = 0; i <= array_size - 1; i++)
		{


			//printf("%d, ", array_of_buckets[i]);

			if(array_of_buckets[i] == -1)
				bucket ++;

			if(bucket == current && start_set == false)
			{
				start = i;
				if(start != 0)
					start ++;

				start_set = true;
			}//end if

			if(bucket == current + 1 && finish_set == false)
			{

				finish = i - 1;

				finish_set = true;


			}//end if

		}//end for i

		//finish = finish - 2;

		printf("current: %d, start: %d, finish: %d\n", current, start, finish);

		if(start < finish)
		{
			if(current == 1)
			{printf("before: ");
				print_array_device(array_of_buckets, array_size);
			}
			bubble_sort(array_of_buckets, size, start, finish);
			if(current == 1)
			{printf("after: ");
				print_array_device(array_of_buckets, array_size);
			}}//end if

		//print_array_device(array_of_buckets, array_size);


		//print_array_device(array_of_buckets[current], bucket_counts[current]);

		//bubble_sort(array_of_buckets[current], bucket_counts[current]);



	}//end function


int find_max_significant_digit(int * array, int size)
{






	return 0;

}//end function


void print_array_(int * host_array, int size)
{

	for(int i = 0; i <= size - 1; i ++)
	{

		printf("%d, ", host_array[i]);

	}//end for i

	printf("\n");

}//end function

void unit_test()
{

int *array;

int size;

int seed;

int bucket;

int max_value;

int * host_array;

int ** array_of_buckets;

int *bucket_counts;

int *array_of_buckets_1D;

for(int z = 0; z <= 10 - 1; z ++)
{

printf("z: %d\n", z);

//int *array;

size = 10;

seed = 10;

	array = makeRandArray( size, seed );
printf("array created\n");

	host_array = (int*)malloc(size * 4);

	for(int i =0; i <= size - 1; i ++)
	{

		host_array[i] = array[i];

	}//end for i

	print_array(array, size);

	printf("host_array\n");

	print_array(host_array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	int * cuda_array;

	hipMalloc(&cuda_array, size * 4);

	hipMemcpy(cuda_array, host_array, size * 4, hipMemcpyHostToDevice);
	
	
	int total_threads = (size / 10);

if(total_threads > 48)
{

	total_threads = 48;

}//end if

	int diameter = sqrt(total_threads) + 1;

	printf("total threads: %d, diameter: %d\n", total_threads, diameter);

	int number_of_digits = 32;

	int threads_on_a_side = diameter / 2;

	printf("threads_on_a_side: %d\n", threads_on_a_side);

	int blocks_on_a_side = (diameter / threads_on_a_side) + 1;

	printf("blocks_on_a_side: %d\n", blocks_on_a_side);

	int number_of_threads = pow(blocks_on_a_side * threads_on_a_side, 2);
	int number_of_buckets = number_of_threads;

	printf("number of threads: %d, buckets: %d\n", number_of_threads, number_of_buckets);

	dim3 threadsPerBlock(threads_on_a_side, threads_on_a_side);

	dim3 numBlocks(blocks_on_a_side, blocks_on_a_side);
	
	array_of_buckets = new int*[number_of_buckets];

	bucket_counts = new int[number_of_buckets];


	int bucket_memory = 100000;

	for(int i = 0; i <= number_of_buckets - 1; i ++)
	{

		array_of_buckets[i] = new int[bucket_memory];


	}//end for i

	max_value = 0;


	for(int i = 0; i <= size - 1; i ++)
	{


		if(array[i] > max_value)
			max_value = array[i];


	}//end for i

	printf("max: %d\n", max_value);

	for(int i = 0; i <= size - 1; i ++)
	{

printf("i: %d\n", i);

printf("array[i]: %d\n", array[i]);

printf("bucket: %d\n", bucket);

printf("number of buckets: %d\n", number_of_buckets);

printf("value: %.2f\n", ((double)array[i] / (double)(max_value + 1)) * number_of_buckets);

		bucket = ((double)array[i] / (double)(max_value + 1)) * number_of_buckets;

		printf("array[i]: %d, bucket: %d, ", array[i], bucket);

		printf("array[i] / max_value: %f, ", (double)array[i] / (double)(max_value + 1)); 

		array_of_buckets[bucket][bucket_counts[bucket]] = array[i]; 

		printf("value_in_array: %d, ", array_of_buckets[bucket][bucket_counts[bucket]]);

		bucket_counts[bucket] ++;

		printf("bucket count: %d, %d\n", 
				bucket_counts[bucket], 
				array_of_buckets[bucket][bucket_counts[bucket] - 1]);


	}//end for i


	int * cuda_bucket_counts;

	hipMalloc(&cuda_bucket_counts, number_of_buckets * 4);

	hipMemcpy(cuda_bucket_counts, bucket_counts, number_of_buckets * 4, hipMemcpyHostToDevice);

size_t array_of_buckets_1D_size = size * 2;

	array_of_buckets_1D = new int[array_of_buckets_1D_size];

	int iter = 0;

	for(int i = 0; i <= number_of_buckets - 1; i++)
	{

		for(int j = 0; j <= bucket_counts[i] - 1; j++)
		{


			array_of_buckets_1D[iter] = array_of_buckets[i][j];

			iter ++;

		}//end for j


		array_of_buckets_1D[iter] = -1;

		iter ++;


	}//end for i

	for(int i = 0; i <= iter - 1; i ++)
	{

		printf("%d, ", array_of_buckets_1D[i]);


	}//end for i


	int * cuda_array_of_buckets;


	hipMalloc(&cuda_array_of_buckets, size * 2 * 4);

	hipMemcpy(cuda_array_of_buckets, array_of_buckets_1D, size * 2 * 4
			, hipMemcpyHostToDevice);

	matavgKernel <<< numBlocks, threadsPerBlock >>> 
		(cuda_array, size, blocks_on_a_side, 
		 number_of_threads, cuda_array_of_buckets, iter, cuda_bucket_counts); 

hipMemcpy(array_of_buckets_1D, cuda_array_of_buckets, size * 2 * 4, hipMemcpyDeviceToHost);

hipFree(cuda_array_of_buckets);

hipFree(cuda_bucket_counts);

printf("after sort(unit test):\n");

print_array(array_of_buckets_1D, iter);

	hipMemcpy(host_array, cuda_array, size * 4, hipMemcpyDeviceToHost);

	hipFree(cuda_array);

int j = 0;

for(int i = 0; i <= iter - 1; i ++)
{

if(array_of_buckets_1D[i] != -1)
{

	host_array[j] = array_of_buckets_1D[i];

	j++;

}//end if

}//end for i

	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );

		//print_array_(host_array, size);

printf("final array: ");

		print_array_(host_array, size);

free(array);

free(host_array);

fflush(stdout);

//free(array_of_buckets);

//free(bucket_counts);

free(array_of_buckets_1D);

		}//end for

//free(array);

//free(host_array);

}//end function


int main( int argc, char* argv[] ) {
	int * array; // the poitner to the array of rands 
	int size, seed; // values for the size of the array 
	bool printSorted = false;
	// and the seed for generating
	// random numbers
	// check the command line args
	if( argc < 3 ){
		std::cerr << "usage: "
			<< argv[0]
			<< " [amount of random nums to generate] [seed value for rand]" << " [1 to print sorted array, 0 otherwise]"
			<< std::endl;
		exit( -1 ); }
	// convert cstrings to ints
	{
		std::stringstream ss1( argv[1] );
		ss1 >> size;
	} {
		std::stringstream ss1( argv[2] ); 
		ss1 >> seed; }

//unit_test();

	array = makeRandArray( size, seed );

	int * host_array = (int*)malloc(size * 10);

	for(int i =0; i <= size - 1; i ++)
	{

		host_array[i] = array[i];

	}//end for i

	print_array(array, size);

	printf("host_array\n");

	print_array(host_array, size);

	hipEvent_t startTotal, stopTotal; float timeTotal; hipEventCreate(&startTotal); hipEventCreate(&stopTotal); hipEventRecord( startTotal, 0 );

	/////////////////////////////////////////////////////////////////////
	///////////////////////  YOUR CODE HERE       ///////////////////////
	/////////////////////////////////////////////////////////////////////

	int * cuda_array;

	hipMalloc(&cuda_array, size * 4);

	hipMemcpy(cuda_array, host_array, size * 4, hipMemcpyHostToDevice);

	int total_threads = (size / 10);

if(total_threads > 48)
total_threads = 48;

	int diameter = sqrt(total_threads) + 1;

	printf("total threads: %d, diameter: %d\n", total_threads, diameter);

	int number_of_digits = 32;

	int threads_on_a_side = diameter / 2;

	printf("threads_on_a_side: %d\n", threads_on_a_side);

	int blocks_on_a_side = (diameter / threads_on_a_side) + 1;

	printf("blocks_on_a_side: %d\n", blocks_on_a_side);

	int number_of_threads = pow(blocks_on_a_side * threads_on_a_side, 2);
	int number_of_buckets = number_of_threads;

	printf("number of threads: %d, buckets: %d\n", number_of_threads, number_of_buckets);

	dim3 threadsPerBlock(threads_on_a_side, threads_on_a_side);

	dim3 numBlocks(blocks_on_a_side, blocks_on_a_side);

	int ** array_of_buckets = new int*[number_of_buckets];

	int *bucket_counts = new int[number_of_buckets];


	int bucket_memory = 10000;

	for(int i = 0; i <= number_of_buckets - 1; i ++)
	{

		array_of_buckets[i] = new int[bucket_memory];


	}//end for i

	int max_value = 0;


	for(int i = 0; i <= size - 1; i ++)
	{


		if(array[i] > max_value)
			max_value = array[i];


	}//end for i

	printf("max: %d\n", max_value);

	for(int i = 0; i <= size - 1; i ++)
	{

		int bucket = ((double)array[i] / (double)(max_value + 1)) * number_of_buckets;

		printf("array[i]: %d, bucket: %d, ", array[i], bucket);

		printf("array[i] / max_value: %f, ", (double)array[i] / (double)(max_value + 1)); 

		array_of_buckets[bucket][bucket_counts[bucket]] = array[i]; 

		printf("value_in_array: %d, ", array_of_buckets[bucket][bucket_counts[bucket]]);

		bucket_counts[bucket] ++;

		printf("bucket count: %d, %d\n", 
				bucket_counts[bucket], 
				array_of_buckets[bucket][bucket_counts[bucket] - 1]);


	}//end for i


	int * cuda_bucket_counts;

	hipMalloc(&cuda_bucket_counts, number_of_buckets * 4);

	hipMemcpy(cuda_bucket_counts, bucket_counts, number_of_buckets * 4, hipMemcpyHostToDevice);

size_t array_of_buckets_1D_size = size * 10;

	int * array_of_buckets_1D = new int[array_of_buckets_1D_size];

	int iter = 0;

	for(int i = 0; i <= number_of_buckets - 1; i++)
	{

		for(int j = 0; j <= bucket_counts[i] - 1; j++)
		{


			array_of_buckets_1D[iter] = array_of_buckets[i][j];

			iter ++;

		}//end for j


		array_of_buckets_1D[iter] = -1;

		iter ++;


	}//end for i

	for(int i = 0; i <= iter - 1; i ++)
	{

		printf("%d, ", array_of_buckets_1D[i]);


	}//end for i


	int * cuda_array_of_buckets;


	hipMalloc(&cuda_array_of_buckets, array_of_buckets_1D_size);

	hipMemcpy(cuda_array_of_buckets, array_of_buckets_1D, array_of_buckets_1D_size
			, hipMemcpyHostToDevice);

	matavgKernel <<< numBlocks, threadsPerBlock >>> 
		(cuda_array, size, blocks_on_a_side, 
		 number_of_threads, cuda_array_of_buckets, iter, cuda_bucket_counts); 

hipMemcpy(array_of_buckets_1D, cuda_array_of_buckets, array_of_buckets_1D_size, hipMemcpyDeviceToHost);

hipFree(cuda_array_of_buckets);

printf("after sort:\n");

print_array(array_of_buckets_1D, iter);

	hipMemcpy(host_array, cuda_array, size * 4, hipMemcpyDeviceToHost);

	hipFree(cuda_array);

int j = 0;

for(int i = 0; i <= iter - 1; i ++)
{

if(array_of_buckets_1D[i] != -1)
{

	host_array[j] = array_of_buckets_1D[i];

	j++;

}//end if

}//end for i

	//https://stackoverflow.com/questions/6419700/way-to-verify-kernel-was-executed-in-cuda

	/***********************************
	 *
	 Stop and destroy the cuda timer
	 **********************************/
	hipEventRecord( stopTotal, 0 );
	hipEventSynchronize( stopTotal );
	hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
	hipEventDestroy( startTotal );
	hipEventDestroy( stopTotal );
	/***********************************
	  end of cuda timer destruction
	 **********************************/
	std::cerr << "Total time in seconds: "
		<< timeTotal / 1000.0 << std::endl;
	printSorted = true;

for(int i = 1; i <= size - 1; i ++)
{

assert(host_array[i] > host_array[i - 1]); 

}//end for i

for(int i = 0; i <= size - 1; i ++)
{
	int missing_number = 1;

printf("checking: %d, ", array[i]);

	for(int j = 0; j <= size - 1; j ++)
	{
	
	if(array[i] == host_array[j])
	{
	
printf("FOUND\n");

	missing_number = 0;
	
	}//end if
	
	}//end for j

	assert(missing_number == 0);

}//end for i


	if( printSorted ){

		print_array_(host_array, size);

		///////////////////////////////////////////////
		/// Your code to print the sorted array here //
		///////////////////////////////////////////////
	}//end if 


}//end main
